#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void init_kernel(float *data, size_t size) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        data[index] = index * 1.0f;
    }
}

int main(int argc, char *argv[]) {
    // const int num_elements = 1024;
    // const int num_elements = 1879048192;
    const size_t num_elements = 18790481920;
    const size_t size = num_elements * sizeof(float);

    // Set CUDA device to 0
    hipSetDevice(atoi(argv[1]));

    // Allocate and initialize memory on the GPU
    float *gpu_data;
    hipMalloc(&gpu_data, size);

    hipDeviceSynchronize();
    std::cout << "Buffer allocated on GPU " << atoi(argv[1]) << std::endl;

    init_kernel<<<(num_elements + 255) / 256, 256>>>(gpu_data, num_elements);

    hipDeviceSynchronize();
    std::cout << "init_kernel synced" << std::endl;

    // Allocate pinned host memory
    float *cpu_data;
    hipHostAlloc(&cpu_data, size, hipHostMallocDefault);

    hipDeviceSynchronize();
    std::cout << "host memory allocated" << std::endl;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Copy data from GPU to pinned host memory
    hipMemcpy(cpu_data, gpu_data, size, hipMemcpyDeviceToHost);

    // Record the stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Time taken for cudaMemcpy: " << milliseconds << " milliseconds. For GPU " << argv[1] << std::endl;

    // Free memory
    hipFree(gpu_data);
    hipHostFree(cpu_data);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}